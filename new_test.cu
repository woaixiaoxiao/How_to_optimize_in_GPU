#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor
                  << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB"
                  << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << std::endl;
    }

    return 0;
}