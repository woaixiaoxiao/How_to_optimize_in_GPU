#include <iostream>
#include <hip/hip_runtime.h>

__global__ void test()
{
    int tid = threadIdx.x;
    auto test_lambda = [&] __device__() { printf("%d\n", tid); };
    test_lambda();
}

int main()
{
    test<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}